#include "hip/hip_runtime.h"
/*
Copyright (C) 2009 Scott Grauer-Gray, Chandra Kambhamettu, and Kannappan Palaniappan

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 2 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307 USA
*/

//This kernal is used to filter the image with the given filter in the vertical and horizontal directions


#include "kernalFilterHeader.cuh"

//checks if the current point is within the image bounds
__device__ bool withinImageBoundsFilter(int xVal, int yVal, int width, int height)
{
	return ((xVal >= 0) && (xVal < width) && (yVal >= 0) && (yVal < height));
}


//kernal to convert the unsigned int pixels to float pixels in an image when
//smoothing is not desired but the pixels need to be converted to floats
//the input image is stored as unsigned ints in the texture imagePixelsUnsignedIntToFilterTexture
//output filtered image stored in floatImagePixels
__global__ void convertUnsignedIntImageToFloat(float* floatImagePixels)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImageConstFilt, heightImageConstFilt))
	{
		//retrieve the float-value of the unsigned int pixel value at the current location
		float floatPixelVal = 1.0f*tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, yVal*widthImageConstFilt + xVal);

		floatImagePixels[yVal*widthImageConstFilt + xVal] = floatPixelVal;
	}
}


//kernal to apply a horizontal filter on each pixel of the image in parallel
//input image stored in texture imagePixelsFloatToFilterTexture
//output filtered image stored in filteredImagePixels
__global__ void filterFloatImageAcross(float* filteredImagePixels)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImageConstFilt, heightImageConstFilt))
	{

		float filteredPixelVal = imageFilterConst[0]*tex1Dfetch(imagePixelsFloatToFilterTexture, yVal*widthImageConstFilt + xVal) ;


		for (int i = 1; i < sizeFilterConst; i++) 
		{
			filteredPixelVal += imageFilterConst[i] * (tex1Dfetch(imagePixelsFloatToFilterTexture, yVal*widthImageConstFilt + max(xVal-i, 0)) 
				+ tex1Dfetch(imagePixelsFloatToFilterTexture, yVal*widthImageConstFilt + min(xVal+i, widthImageConstFilt-1))); 
		}

		filteredImagePixels[yVal*widthImageConstFilt + xVal] = filteredPixelVal;
	}
}


//kernal to apply a vertical filter on each pixel of the image in parallel
//input image stored in texture imagePixelsFloatToFilterTexture
//output filtered image stored in filteredImagePixels
__global__ void filterFloatImageVertical(float* filteredImagePixels)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImageConstFilt, heightImageConstFilt))
	{

		float filteredPixelVal = imageFilterConst[0]*tex1Dfetch(imagePixelsFloatToFilterTexture, yVal*widthImageConstFilt + xVal);


		for (int i = 1; i < sizeFilterConst; i++) {
			filteredPixelVal += imageFilterConst[i] * (tex1Dfetch(imagePixelsFloatToFilterTexture, max(yVal-i, 0)*widthImageConstFilt + xVal) 
				+ tex1Dfetch(imagePixelsFloatToFilterTexture, min(yVal+i, heightImageConstFilt-1)*widthImageConstFilt + xVal)); 
		}

		filteredImagePixels[yVal*widthImageConstFilt + xVal] = filteredPixelVal;
	}
}

//kernal to apply a horizontal filter on each pixel of the image in parallel
//the input image is stored as unsigned ints in the texture imagePixelsUnsignedIntToFilterTexture
//the output filtered image is returned as an array of floats
__global__ void filterUnsignedIntImageAcross(float* filteredImagePixels)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImageConstFilt, heightImageConstFilt))
	{

		float filteredPixelVal = imageFilterConst[0]*tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, yVal*widthImageConstFilt + xVal) ;


		for (int i = 1; i < sizeFilterConst; i++) {
			filteredPixelVal += imageFilterConst[i] * (tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, yVal*widthImageConstFilt + max(xVal-i, 0)) 
				+ tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, yVal*widthImageConstFilt + min(xVal+i, widthImageConstFilt-1))); 
		}

		filteredImagePixels[yVal*widthImageConstFilt + xVal] = filteredPixelVal;
	}
}


//kernal to apply a vertical filter on each pixel of the image in parallel
//the input image is stored as unsigned ints in the texture imagePixelsUnsignedIntToFilterTexture
//the output filtered image is returned as an array of floats
__global__ void filterUnsignedIntImageVertical(float* filteredImagePixels)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xVal = bx * BLOCK_SIZE_WIDTH_FILTER_IMAGES + tx;
	int yVal = by * BLOCK_SIZE_HEIGHT_FILTER_IMAGES + ty;

	//make sure that (xVal, yVal) is within image bounds
	if (withinImageBoundsFilter(xVal, yVal, widthImageConstFilt, heightImageConstFilt))
	{

		float filteredPixelVal = imageFilterConst[0]*tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, yVal*widthImageConstFilt + xVal);


		for (int i = 1; i < sizeFilterConst; i++) {
			filteredPixelVal += imageFilterConst[i] * (tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, max(yVal-i, 0)*widthImageConstFilt + xVal) 
				+ tex1Dfetch(imagePixelsUnsignedIntToFilterTexture, min(yVal+i, heightImageConstFilt-1)*widthImageConstFilt + xVal)); 
		}

		filteredImagePixels[yVal*widthImageConstFilt + xVal] = filteredPixelVal;
	}
}
